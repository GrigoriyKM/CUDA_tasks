#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <time.h>

const int N = 1000000000; // 10^9

// Ядро для инициализации массива
__global__ void initializeArray(float *arr)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        arr[i] = __sinf((i % 360) * M_PI / 180);
    }
}

double calcError(float *hostArr, int arraySize)
{
    double err = 0;
    for (int i = 0; i < arraySize; i++)
    {
        err += abs(sin((i % 360) * M_PI / 180) - hostArr[i]);
    }
    return err / arraySize;
}

void checkCudaError(hipError_t status)
{
    if (status != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}

int main()
{
    // Время начала выполнения
    clock_t start = clock();
    int device = 1;
    hipSetDevice(device);
    // Выделение памяти на GPU для массива
    float *d_arr;
    checkCudaError(hipMalloc(&d_arr, N * sizeof(float)));

    dim3 blockSize = dim3(256);
    dim3 numBlocks = dim3((N + blockSize.x - 1) / blockSize.x);
    // Запуск ядра для инициализации массива
    initializeArray<<<numBlocks, blockSize>>>(d_arr);

    // Выделение памяти на CPU для массива
    float *h_arr = (float *)malloc(N * sizeof(float));

    // Копирование массива с GPU на CPU
    checkCudaError(hipMemcpy(h_arr, d_arr, N * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaError(hipDeviceSynchronize());

    // Расчет ошибки
    printf("Ошибка (__sinf) = %0.10f \n", calcError(h_arr, N));

    // Освобождение памяти на GPU
    hipFree(d_arr);

    // Освобождение памяти на CPU
    free(h_arr);

    // Время окончания выполнения
    clock_t end = clock();

    // Вывод времени выполнения
    printf("Время выполнения: %0.5f секунд \n", (end - start) / CLOCKS_PER_SEC);

    return 0;
};