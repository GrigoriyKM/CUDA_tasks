#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <time.h>

const int N = 1000000000; // 10^9

// Ядро для инициализации массива
__global__ void initializeArray(double *arr)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        arr[i] = __sinf((i % 360) * M_PI / 180);
    }
}

double calcError(double *hostArr, int arraySize)
{
    double err = 0;
    for (int i = 0; i < arraySize; i++)
    {
        err += abs(sin((i % 360) * M_PI / 180) - hostArr[i]);
    }
    return err / arraySize;
}

void checkCudaError(hipError_t status)
{
    if (status != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}

int main()
{

    int device = 1;
    hipSetDevice(device);
    // Выделение памяти на GPU для массива
    double *d_arr;
    checkCudaError(hipMalloc(&d_arr, N * sizeof(double)));

    dim3 blockSize = dim3(256);
    dim3 numBlocks = dim3((N + blockSize.x - 1) / blockSize.x);

    // Время начала выполнения
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // Запуск ядра для инициализации массива
    initializeArray<<<numBlocks, blockSize>>>(d_arr);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Выделение памяти на CPU для массива
    double *h_arr = (double *)malloc(N * sizeof(double));

    // Копирование массива с GPU на CPU
    checkCudaError(hipMemcpy(h_arr, d_arr, N * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaError(hipDeviceSynchronize());

    // Расчет ошибки
    printf("Ошибка (__sinf) = %0.10f \n", calcError(h_arr, N));

    // Вывод времени выполнения
    printf("Время выполнения: %f мсек \n", milliseconds);

    // Освобождение памяти на GPU
    hipFree(d_arr);

    // Освобождение памяти на CPU
    free(h_arr);

    return 0;
};